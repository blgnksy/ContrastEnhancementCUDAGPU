#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include <assert.h>
#include <math.h>
#include <windows.h>

// CUDA error checking Macro.
#define CUDA_CALL(x,y) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}\
  else{printf("CUDA Success at %d. (%s)\n",__LINE__,y); }}

//Global
#define DIM 256

// Function Prototypes.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray);

__global__ void
MinMaxKernel(Npp8u * pSrc_Dev, Npp8u * pMin_Dev, Npp8u * pMax_Dev);

__global__ void
SubtractKernel(Npp8u * pDst_Dev, Npp8u * pSrc_Dev, Npp8u nMin_Dev);

__global__ void
MultiplyKernel(Npp8u * pDst_Dev, Npp8u nConstant, int normalizer);

void StartCounter();

double GetCounter();


int
main(int argc, char ** argv)
{
	// Host parameter declarations.	
	Npp8u * pSrc_Host, *pDst_Host;
	int   nWidth, nHeight, nMaxGray;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	pSrc_Host = LoadPGM("C:\\Users\\blgnksy\\source\\repos\\CudaAssignment2\\ColorEnhancement\\lena_before.pgm", nWidth, nHeight, nMaxGray);
	pDst_Host = new Npp8u[nWidth * nHeight];

	// Device parameter declarations.
	Npp8u	 * pSrc_Dev, *pDst_Dev;
	Npp8u    * pMin_Dev, *pMax_Dev;
	Npp8u    nMin_Host[DIM], nMax_Host[DIM];
	int		 nSrcStep_Dev, nDstStep_Dev;

	//Start Counter.
	hipEvent_t start, stop;
	float elapsed_time_ms;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Allocate and Copy the Device Variables
	pSrc_Dev = nppiMalloc_8u_C1(nWidth, nHeight, &nSrcStep_Dev);
	pDst_Dev = nppiMalloc_8u_C1(nWidth, nHeight, &nDstStep_Dev);

	// Device variables are copied to device. 
	CUDA_CALL(hipMalloc(&pMin_Dev, DIM * sizeof(Npp8u)), "Memory allocated.");
	CUDA_CALL(hipMalloc(&pMax_Dev, DIM * sizeof(Npp8u)), "Memory allocated.");
	CUDA_CALL(hipMemcpy(pSrc_Dev, pSrc_Host, nWidth * nHeight * sizeof(Npp8u), hipMemcpyHostToDevice), "Memory copied.(HostToDevice)");

	/*
	Defining Kernel Execution Paramaters.
	I defined two different block size to be able to find global minimum. During the First Max and Min kernels execution, they are only
	be able to find local minimum.
	*/
	dim3 dimGrid(nWidth);
	dim3 dimBlock2(nWidth / 2);
	dim3 dimBlock1(nWidth);

	//Minimum kernel and Maximum kernels are independent. So no need to put them into same stream.
	size_t sharedMemSize = nHeight *  nWidth * sizeof(Npp8u);

	// One kernel for both min and max.
	MinMaxKernel << <dimGrid, dimBlock2, sharedMemSize >> > (pSrc_Dev, pMin_Dev, pMax_Dev);
	MinMaxKernel << <1, dimBlock1, sharedMemSize >> > (pMin_Dev, pMin_Dev, pMax_Dev);

	// Minimum and maximum values are copied to host.
	CUDA_CALL(hipMemcpy(&nMin_Host, pMin_Dev, sizeof(Npp8u), hipMemcpyDeviceToHost), "Memory copied.(DeviceToHost)");
	CUDA_CALL(hipMemcpy(&nMax_Host, pMax_Dev, sizeof(Npp8u), hipMemcpyDeviceToHost), "Memory copied.(DeviceToHost)");

	//Just for control.
	assert(nMin_Host[0] = 92);
	assert(nMax_Host[0] = 202);

	// Subtracting the minimum
	SubtractKernel << <dimGrid, dimBlock1 >> > (pDst_Dev, pSrc_Dev, nMin_Host[0]);

	// Provided code from Original work.
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (nMax_Host[0] - nMin_Host[0]) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}
	Npp8u nConstant = static_cast<Npp8u>(255.0f / (nMax_Host[0] - nMin_Host[0]) * (nPower / 2));

	// CUDA Kernel doesn't support these calculation. So that I calculated it outside the kernel. 
	int normalizer = pow(2, (nScaleFactor - 1));

	// Multiply constant and 
	MultiplyKernel << <dimGrid, dimBlock1 >> > (pDst_Dev, nConstant, normalizer);

	CUDA_CALL(hipMemcpy(pDst_Host, pDst_Dev, nWidth * nHeight * sizeof(Npp8u), hipMemcpyDeviceToHost), "Memory copied.(DeviceToHost)");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("Time to calculate results(GPU Time): %f ms.\n", elapsed_time_ms);

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM("C:\\Users\\blgnksy\\source\\repos\\CudaAssignment2\\ColorEnhancement\\lena_after_GPU.pgm", pDst_Host, nWidth, nHeight, nMaxGray);

	// Clean up.
	delete[] pSrc_Host;
	delete[] pDst_Host;

	nppiFree(pSrc_Dev);
	nppiFree(pDst_Dev);
	CUDA_CALL(hipFree(pMin_Dev), "Memory Freed.");
	CUDA_CALL(hipFree(pMax_Dev), "Memory Freed.");
	printf("All done. Press Any Key to Continue...");
	getchar();
	return 0;
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	Npp8u * pSrc_Host = new Npp8u[nWidth * nHeight];
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			pSrc_Host[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return pSrc_Host;
}

// Write PGM image.
void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "w+");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = "# Created by CUDA Assignment II";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(pDst_Host[i*nWidth + j], fOutput);
	fclose(fOutput);
}

//
__global__ void
MinMaxKernel(Npp8u * pSrc_Dev, Npp8u * pMin_Dev, Npp8u * pMax_Dev)
{
	extern __shared__ Npp8u sMin[];
	extern __shared__ Npp8u sMax[];
	unsigned int tid = threadIdx.x;
	unsigned int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	if (gid < 512)
	{
		if (pSrc_Dev[gid] > pSrc_Dev[gid + blockDim.x])
		{
			sMin[tid] = pSrc_Dev[gid + blockDim.x];
			sMax[tid] = pSrc_Dev[gid];
		}
		else
		{
			sMin[tid] = pSrc_Dev[gid];
			sMax[tid] = pSrc_Dev[gid + blockDim.x];
		}
		__syncthreads();
	}
	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (tid < s)
			if (sMin[tid] > sMin[tid + s]) sMin[tid] = sMin[tid + s];
		if (sMax[tid] < sMax[tid + s]) sMax[tid] = sMax[tid + s];
		__syncthreads();
	}
	if (tid < 32)
	{
		if (sMin[tid] > sMin[tid + 32]) sMin[tid] = sMin[tid + 32];
		if (sMin[tid] > sMin[tid + 16]) sMin[tid] = sMin[tid + 16];
		if (sMin[tid] > sMin[tid + 8]) sMin[tid] = sMin[tid + 8];
		if (sMin[tid] > sMin[tid + 4]) sMin[tid] = sMin[tid + 4];
		if (sMin[tid] > sMin[tid + 2]) sMin[tid] = sMin[tid + 2];
		if (sMin[tid] > sMin[tid + 1]) sMin[tid] = sMin[tid + 1];

		if (sMax[tid] < sMax[tid + 32]) sMax[tid] = sMax[tid + 32];
		if (sMax[tid] < sMax[tid + 16]) sMax[tid] = sMax[tid + 16];
		if (sMax[tid] < sMax[tid + 8]) sMax[tid] = sMax[tid + 8];
		if (sMax[tid] < sMax[tid + 4]) sMax[tid] = sMax[tid + 4];
		if (sMax[tid] < sMax[tid + 2]) sMax[tid] = sMax[tid + 2];
		if (sMax[tid] < sMax[tid + 1]) sMax[tid] = sMax[tid + 1];
	}
	if (tid == 0)
	{
		pMin_Dev[blockIdx.x] = sMin[0];
		pMax_Dev[blockIdx.x] = sMax[0];
	}
}

// Subtract Min from Source and set it to Destination
__global__ void
SubtractKernel(Npp8u * pDst_Dev, Npp8u * pSrc_Dev, Npp8u nMin_Dev)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	pDst_Dev[i] = pSrc_Dev[i] - nMin_Dev;
}

// multiply by nConstant and divide by 2 ^ nScaleFactor-1
__global__ void
MultiplyKernel(Npp8u * pDst_Dev, Npp8u nConstant, int normalizer)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	pDst_Dev[i] = static_cast<Npp8u>(pDst_Dev[i] * nConstant / normalizer);
}